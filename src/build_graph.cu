#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "onecut_kernel.h"

int updiv(int x, int y) { return (x + y - 1) / y; }

__device__ float sigma_square = 0;

__device__ void convertToRGB(int pixel_value, int *r, int *g, int *b) {
  *b = pixel_value & 255;
  pixel_value >>= 8;
  *g = pixel_value & 255;
  pixel_value >>= 8;
  *r = pixel_value & 255;
}

__device__ int Di(int pixel_p, int pixel_q) {
  int p_r, p_g, p_b;
  int q_r, q_g, q_b;
  convertToRGB(pixel_p, &p_r, &p_g, &p_b);
  convertToRGB(pixel_q, &q_r, &q_g, &q_b);
  return (p_r - q_r) * (p_r - q_r) + (p_g - q_g) * (p_g - q_g) +
         (p_b - q_b) * (p_b - q_b);
}

__device__ void warpReduce(volatile int *sigma_sum, int tid, int block_dim_x) {
  sigma_sum[tid] += tid + 32 >= block_dim_x ? 0 : sigma_sum[tid + 32];
  sigma_sum[tid] += tid + 16 >= block_dim_x ? 0 : sigma_sum[tid + 16];
  sigma_sum[tid] += tid + 8 >= block_dim_x ? 0 : sigma_sum[tid + 8];
  sigma_sum[tid] += tid + 4 >= block_dim_x ? 0 : sigma_sum[tid + 4];
  sigma_sum[tid] += tid + 2 >= block_dim_x ? 0 : sigma_sum[tid + 2];
  sigma_sum[tid] += tid + 1 >= block_dim_x ? 0 : sigma_sum[tid + 1];
}

__global__ void computeSigmaSquareSum(int img_width, int img_height,
                                      const int *__restrict__ src_img) {
  extern __shared__ int sigma_sum[];

  int tid = threadIdx.x;
  int block_id = blockIdx.y * gridDim.x + blockIdx.x;
  int thread_id = block_id * blockDim.x + threadIdx.x;

  int img_size = img_width * img_height;
  sigma_sum[tid] = 0;
  if (thread_id * 2 < img_size) {
    int p_idx = thread_id * 2;
    int pixel_p = src_img[p_idx];
    int p_x = p_idx / img_width;

    if (p_x + 1 < img_height) {  // p-down
      sigma_sum[tid] += Di(pixel_p, src_img[p_idx + img_width]);
    }

    if (p_idx + 1 < img_size) {  // q is valid
      int pixel_q = src_img[p_idx + 1];
      int q_x = (p_idx + 1) / img_width, q_y = (p_idx + 1) % img_width;

      if (p_x == q_x) {  // p-right
        sigma_sum[tid] += Di(pixel_p, pixel_q);
      }

      if (q_y + 1 < img_width) {  // q-right
        sigma_sum[tid] += Di(pixel_q, src_img[p_idx + 2]);
      }

      if (q_x + 1 < img_height) {  // q-down
        sigma_sum[tid] += Di(pixel_q, src_img[p_idx + 1 + img_width]);
      }
    }
  }

  __syncthreads();
  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) {
      sigma_sum[tid] += sigma_sum[tid + s];
    }
    __syncthreads();
  }

  if (tid < 32) {
    warpReduce(sigma_sum, tid, blockDim.x);
  }

  if (tid == 0) {
    atomicAdd(&sigma_square, sigma_sum[0]);
  }
}

__host__ void computeSigmaSquare(int img_height, int img_width,
                                 int *d_src_img) {
  int img_size = img_height * img_width;

  // compute the square of sigma
  dim3 grid(1, 1, 1), block(1024, 1, 1);
  if (img_size < 1024 * 2) {
    block.x = updiv(img_size, 2);
  } else {
    grid.x = updiv(img_size, 1024 * 2);
  }

  computeSigmaSquareSum<<<grid, block, block.x * sizeof(int)>>>(
      img_width, img_height, d_src_img);

  int N = (img_height - 1) * img_width + (img_width - 1) * img_height;
  float h_sigma_square;
  hipMemcpyFromSymbol((void *)&h_sigma_square, sigma_square, sizeof(float));

  h_sigma_square /= N;

  // printf("sigma_square = %lf\n", h_sigma_square);

  hipMemcpyToSymbol(HIP_SYMBOL(sigma_square), (void *)&h_sigma_square, sizeof(float));
}

__device__ float gaussian(int di, float lambda, float sigma_square) {
  return lambda * exp(-di / (2 * sigma_square));
}

__device__ int getColorBinIdx(int pixel_value, int color_bin_size) {
  int r, g, b;
  convertToRGB(pixel_value, &r, &g, &b);

  int per_bin_channel = 256 / color_bin_size;
  return (r / color_bin_size) * per_bin_channel * per_bin_channel +
         (g / color_bin_size) * per_bin_channel + (b / color_bin_size);
}

__global__ void computeEdges(float lambda, float beta, float *edges,
                             int img_width, int img_height, int color_bin_size,
                             const int *__restrict__ src_img,
                             const int *__restrict__ mask_img) {
  int block_id = blockIdx.y * gridDim.x + blockIdx.x;
  int thread_id = block_id * blockDim.x + threadIdx.x;

  // int color_bin_size = pow(256/color_bin_size, 3);
  // int color_bin_size = 256/color_bin_size;
  int img_size = img_height * img_width;
  int edges_width = 6 + 2 + 2;

  if (thread_id < img_size) {
    int idx = thread_id * (edges_width);
    for (unsigned int i = 0; i < edges_width; ++i) {
      edges[idx + i] = 0;
    }

    // add s-t-links or t-t-links
    int seed_value = mask_img[thread_id];
    if (seed_value == 255 << 16) {  // s-t-links
      edges[idx] = edges[idx + 8] = 1000;
    } else if (seed_value == 255 << 8) {  // t-t-links
      edges[idx + 1] = 1000;
    }

    // add a-link of color bins
    int color_bin_idx = getColorBinIdx(src_img[thread_id], color_bin_size);
    edges[idx + 5 + 1] = color_bin_idx;
    edges[idx + 5 + 2] = edges[idx + 9] = beta;

    // add n-links
    int pixel_p = src_img[thread_id];
    if (thread_id % img_width + 1 < img_width) {  // right
      edges[idx + 5] = edges[idx + edges_width + 4] =
          gaussian(Di(pixel_p, src_img[thread_id + 1]), lambda, sigma_square);
    }

    if (thread_id + img_width < img_size) {  // down
      edges[idx + 3] = edges[idx + img_width * edges_width + 2] = gaussian(
          Di(pixel_p, src_img[thread_id + img_width]), lambda, sigma_square);
    }
  }
}

__global__ void init(float *res_pixel, float *pixel_flow, int *bin_height,
                     int img_size, int img_height, int img_width, int bin_size) {
  int img_x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x,
      img_y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  int img_idx = __umul24(img_y, img_width) + img_x;
  if(img_idx == 0) {
    bin_height[bin_size] = img_size + bin_size + 2;
  }
  if(img_x < img_width && img_y < img_height) {
    float tmp_res = res_pixel[img_idx * RES_UNIT_SIZE + 8];
    if(tmp_res > EPS) {
      pixel_flow[img_idx] = tmp_res;
      res_pixel[img_idx * RES_UNIT_SIZE + 8] = 0;
      res_pixel[img_idx * RES_UNIT_SIZE + 0] += tmp_res;
    }
  }
}


int *getCutMask(int *src_img, int *mask_img, int img_height, int img_width) {
  float lambda = 1.0;
  float beta = 0.5;
  int color_bin_size = 64;
  // int color_bin_num = 256/color_bin_size;
  int color_bin_num = pow(256 / color_bin_size, 3);

  int img_size = img_height * img_width;
  int img_num_bytes = sizeof(int) * img_size;

  // compute sigma square
  int *d_src_img, *d_mask_img;
  hipMalloc((void **)&d_src_img, img_num_bytes);
  hipMalloc((void **)&d_mask_img, img_num_bytes);
  hipMemcpy(d_src_img, src_img, img_num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_mask_img, mask_img, img_num_bytes, hipMemcpyHostToDevice);

  computeSigmaSquare(img_height, img_width, d_src_img);

  // compute edges
  float *d_edges = NULL;
  int edges_num_bytes = sizeof(float) * img_num_bytes * (6 + 2 + 2);
  hipMalloc((void **)&d_edges, edges_num_bytes);

  dim3 block0(1024, 1, 1), grid0(1, 1, 1);
  if (img_size < 1024) {
    block0.x = img_size;
  } else {
    grid0.x = updiv(img_size, 1024);
  }
  computeEdges<<<grid0, block0>>>(lambda, beta, d_edges, img_width, img_height,
                                color_bin_size, d_src_img, d_mask_img);
  CHECK(hipDeviceSynchronize());

  // initialize data for maxflow
  float *d_bin_flow, *d_pixel_flow, *d_pull_pixel;
  int *d_pixel_height, *d_bin_height;
  bool h_finished, *d_finished;
  float *h_edges = (float *)malloc(edges_num_bytes);
  float *h_pixel_flow = (float *)malloc(img_size * sizeof(float));
  float *h_bin_flow = (float *)malloc((color_bin_num + 1) * sizeof(float));
  int *h_pixel_height = (int *)malloc(img_size * sizeof(int));
  int *h_bin_height = (int *)malloc((color_bin_num + 1) * sizeof(int));

  hipMalloc((void **)&d_bin_flow, (color_bin_num + 1) * sizeof(float));
  hipMalloc((void **)&d_pixel_flow, img_size * sizeof(float));
  hipMalloc((void **)&d_pull_pixel, img_size * sizeof(float));
  hipMalloc((void **)&d_pixel_height, img_size * sizeof(int));
  hipMalloc((void **)&d_bin_height, (color_bin_num + 1) * sizeof(int));
  hipMalloc((void **)&d_finished, sizeof(bool));
  hipMemcpy(h_edges, d_edges, edges_num_bytes, hipMemcpyDeviceToHost);
  hipMemset(d_bin_flow, 0, (color_bin_num + 1) * sizeof(float));
  hipMemset(d_pixel_flow, 0, img_size * sizeof(float));
  hipMemset(d_pull_pixel, 0, img_size * sizeof(float));
  hipMemset(d_pixel_height, 0, img_size * sizeof(int));
  hipMemset(d_bin_height, 0, (color_bin_num + 1) * sizeof(int));


  dim3 block1(32, 32);
  dim3 grid1(updiv(img_width, 32), updiv(img_height, 32));
  init<<<grid1, block1>>>(d_edges, d_pixel_flow, d_bin_height, img_size, img_height, img_width, color_bin_num);
  // maxflow
  dim3 block_bin(1024);
  dim3 grid_bin(updiv(color_bin_num + 1, 1024));
  do {
    h_finished = true;
    hipMemcpy(d_finished, &h_finished, sizeof(bool), hipMemcpyHostToDevice);
    // relabel
    kernel_pixel_relabel<<<grid1, block1,
                           sizeof(int) * (34 * 34 + color_bin_num + 1)>>>(
        d_edges, d_pixel_flow, d_pixel_height, d_bin_height, img_size,
        img_width, img_height, 34 * 34, 34, 34, color_bin_num, d_finished);
    kernel_bin_relabel<<<grid1, block1>>>(
        d_edges, d_pixel_flow, d_bin_flow, d_pixel_height, d_bin_height,
        img_size, img_width, img_height, 34 * 34, 34, 34, color_bin_num,
        d_finished);
    kernel_bin_relabel_rectify<<<grid_bin, block_bin>>>(
        d_bin_height, color_bin_num, d_finished);
    // push & pull
    kernel_pixel_push<<<grid1, block1,
                        34 * 34 * RES_UNIT_SIZE * sizeof(float)>>>(
        d_edges, d_bin_flow, d_pixel_flow, d_pull_pixel, d_pixel_height,
        d_bin_height, img_size, img_width, img_height, 34 * 34, 34, 34,
        color_bin_num);
    kernel_pixel_pull<<<grid1, block1>>>(d_edges, d_pull_pixel, d_pixel_flow,
                                         img_size, img_width, img_height);
    CHECK(hipDeviceSynchronize());
    hipMemcpy(&h_finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);
  } while (!h_finished);

  // bfs
  kernel_bfs_init<<<grid1, block1>>>(d_edges, d_pixel_height, d_bin_height,
                                     img_size, img_width, img_height,
                                     color_bin_num);
  int cur_height = 1;
  do {
    h_finished = true;
    hipMemcpy(d_finished, &h_finished, sizeof(bool), hipMemcpyHostToDevice);
    kernel_pixel_bfs<<<grid1, block1,
                       sizeof(int) * (34 * 34 + color_bin_num + 1)>>>(
        d_edges, d_pixel_height, d_bin_height, img_size, img_width, img_height,
        34 * 34, 34, 34, color_bin_num, cur_height, d_finished);
    kernel_bin_bfs<<<grid1, block1, sizeof(int) * color_bin_num>>>(
        d_edges, d_pixel_height, d_bin_height, img_size, img_width, img_height,
        color_bin_num, cur_height, d_finished);
    hipMemcpy(&h_finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);
    cur_height++;
  } while (!h_finished);

  // segment
  kernel_segment<<<grid1, block1>>>(d_pixel_height, img_size, img_width,
                                    img_height);
  hipMemcpy(h_pixel_height, d_pixel_height, img_size * sizeof(int),
             hipMemcpyDeviceToHost);

  free(h_edges);
  free(h_bin_flow);
  free(h_bin_height);
  free(h_pixel_flow);

  hipFree(d_bin_flow);
  hipFree(d_pixel_flow);
  hipFree(d_pull_pixel);
  hipFree(d_pixel_height);
  hipFree(d_bin_height);
  hipFree(d_edges);
  hipFree(d_src_img);
  hipFree(d_mask_img);

  return h_pixel_height;
}

int main(int argc, char **argv) {
  int img_height, img_width;
//   int img_height = 2, img_width = 3;

  FILE *fp;
  fp = fopen(argv[1], "r");
  fscanf(fp, "%d%d", &img_height, &img_width);

  int *src_img = (int *)malloc(sizeof(int) * img_height * img_width);
  int *mask_img = (int *)malloc(sizeof(int) * img_height * img_width);
  for (int i = 0; i < img_height * img_width; ++i) {
    fscanf(fp, "%d", &src_img[i]);
  }
  for (int i = 0; i < img_height * img_width; ++i) {
    fscanf(fp, "%d", &mask_img[i]);
  }
  fclose(fp);

  int *segment = getCutMask(src_img, mask_img, img_height, img_width);
  for (int j = 0; j < img_width; ++j) {
    for (int i = 0; i < img_height; ++i) {
      printf("%c", segment[i * img_width + j] == 0 ? ' ' : '#');
    }
    printf("\n");
  }
  free(segment);
  return 0;
}
