#include "hip/hip_runtime.h"
#include "common.h"
#include "enhance.h"
#include <stdio.h>
#include <stdlib.h>



__global__ void CLAHE(int * hsi_img, int height, int width)
// the 'tile' size is the same with the block size, 1 block for 9 tile
{
    __shared__ int frq[9*256+9];
    int lt_x = __umul24(blockIdx.x, blockDim.x*3) + threadIdx.x,
      lt_y = __umul24(blockIdx.y, blockDim.y*3) + threadIdx.y;
    int lt_idx = __umul24(lt_y, width) + lt_x;
    int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
    int per_thread = 9;
    if(thread_idx < 256) {
        for (int i = 0; i < per_thread; ++i)
        {
            frq[thread_idx*per_thread+i] = 0;
        }
    }
    if (thread_idx == 0) {
        // printf("%d %d\n", blockIdx.x, blockIdx.y);
        for (int i = 0; i < 9; ++i)
        {
            frq[9*256+i] = 0;
        }
    }
    
    __syncthreads();

    for (int i = 0; i < 3; ++i)
    {
        int tmp_x = lt_x;
        int tmp_y = lt_y + i*TILESIZE;
        int tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            atomicAdd(&frq[(i*3+0)*256+(tex2D(tex2, tmp_x, tmp_y)&0x0000FF)], 1);
        }
        tmp_x = lt_x + TILESIZE;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            atomicAdd(&frq[(i*3+1)*256+(tex2D(tex2, tmp_x, tmp_y)&0x0000FF)], 1);
        }
        tmp_x = lt_x + TILESIZE*2;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            atomicAdd(&frq[(i*3+2)*256+(tex2D(tex2, tmp_x, tmp_y)&0x0000FF)], 1);
        }
    }
    __syncthreads();

    if(thread_idx < 256) {
        for (int i = 0; i < 9; ++i)
        {
            int overflow = (frq[i*256+thread_idx] > THRESHOLD)? frq[i*256+thread_idx] - THRESHOLD : 0;
            frq[i*256+thread_idx] -= overflow;
            atomicAdd(&frq[9*256+i], overflow);
        }
    }
    __syncthreads();

    if(thread_idx < 256) {
        for (int i = 0; i < 9; ++i)
        {
            frq[i*256+thread_idx] += frq[9*256+i]/256;
        }
    }

    __syncthreads();

    for (int i = 0; i < 9; ++i)
    {
        for (int stride = 1; stride < 256; stride <<= 1)
        {
            __syncthreads();
            int val;
            if(thread_idx < 256)
                val = (thread_idx > stride)? frq[i*256+thread_idx-stride]:0;
            __syncthreads();
            if(thread_idx < 256)
                frq[i*256+thread_idx] += val;
        }
    }

    __syncthreads();
    
    for (int i = 0; i < 3; ++i)
    {
        int tmp_x = lt_x;
        int tmp_y = lt_y + i*TILESIZE;
        int tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            hsi_img[tmp_idx] = (tex2D(tex2, tmp_x, tmp_y) & 0xFFFF00) + (1.0*frq[(i*3+0)*256+(tex2D(tex2, tmp_x, tmp_y)&0x0000FF)]/(TILESIZE*TILESIZE))*255;
            // if ((1.0*frq[(i*3+0)*256+(hsi_img[tmp_idx]&0x0000FF)]/(TILESIZE*TILESIZE))*255 > 255) {
            //     printf("==>ERROR!\n");
            // }
        }
        tmp_x = lt_x + TILESIZE;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            hsi_img[tmp_idx] = (tex2D(tex2, tmp_x, tmp_y) & 0xFFFF00) + (1.0*frq[(i*3+1)*256+(tex2D(tex2, tmp_x, tmp_y)&0x0000FF)]/(TILESIZE*TILESIZE))*255;
            // if ((1.0*frq[(i*3+1)*256+(hsi_img[tmp_idx]&0x0000FF)]/(TILESIZE*TILESIZE))*255 > 255) {
            //     printf("==>ERROR!\n");
            // }
        }
        tmp_x = lt_x + TILESIZE*2;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            hsi_img[tmp_idx] = (tex2D(tex2, tmp_x, tmp_y) & 0xFFFF00) + (1.0*frq[(i*3+2)*256+(tex2D(tex2, tmp_x, tmp_y)&0x0000FF)]/(TILESIZE*TILESIZE))*255;
            // if ((1.0*frq[(i*3+2)*256+(hsi_img[tmp_idx]&0x0000FF)]/(TILESIZE*TILESIZE))*255 > 255) {
                // printf("==>ERROR!\n");
            // }
        }
    }
}

// bool compare(int *one, int *two, int img_height, int img_width) {
//     for (int i = 0; i < img_height*img_width; ++i)
//     {
//         if(one[i]&0xFFFF00 != two[i]&0xFFFF00) {
//             return false;
//         }
//     }
//     return true;
// }

int* imgCLAHE(int *src_img, int img_height, int img_width)
{
    int * d_rgb_img, * d_hsi_img;
    int * ret_img;
    int *h_img_one, *h_img_two;
    ret_img = (int*)malloc(img_height*img_width*sizeof(int));
    h_img_one = (int*)malloc(img_height*img_width*sizeof(int));
    h_img_two = (int*)malloc(img_height*img_width*sizeof(int));
    hipMalloc((void**)& d_rgb_img, img_height*img_width*sizeof(int));
    hipMalloc((void**)& d_hsi_img, img_height*img_width*sizeof(int));
    hipMemcpy(d_rgb_img, src_img, img_height*img_width*sizeof(int), hipMemcpyHostToDevice);
    dim3 block(TILESIZE,TILESIZE);
    dim3 grid1(updiv(img_width, TILESIZE), updiv(img_height, TILESIZE));
    dim3 grid2(updiv(img_width, TILESIZE*3), updiv(img_height, TILESIZE*3));

    hipChannelFormatDesc desc1 = hipCreateChannelDesc<int> ();
    hipChannelFormatDesc desc2 = hipCreateChannelDesc<int> ();
    hipBindTexture2D(0, tex1, d_rgb_img, desc1, img_width, img_height, img_width*sizeof(int));
    hipBindTexture2D(0, tex2, d_hsi_img, desc2, img_width, img_height, img_width*sizeof(int));

    RGB2HSI<<<grid1, block>>>(d_rgb_img, d_hsi_img, img_height, img_width);
    CLAHE<<<grid2, block>>>(d_hsi_img, img_height, img_width);
    HSI2RGB<<<grid1, block>>>(d_hsi_img, d_rgb_img, img_height, img_width);
    // if(!compare(h_img_one, h_img_two, img_height, img_width)) {
    //     printf("===> ERROR!HSI matrix changed!\n");
    // }
    
    hipMemcpy(ret_img, d_rgb_img, img_height*img_width*sizeof(int), hipMemcpyDeviceToHost);
    
    hipUnbindTexture(tex1);
    hipUnbindTexture(tex2);
    hipFree(d_rgb_img);
    hipFree(d_hsi_img);
    free(h_img_one);
    free(h_img_two);
    return ret_img;
}

int main(int argc, char **argv) {
  int img_height, img_width;

  FILE *fp;
  fp = fopen(argv[1], "r");
  fscanf(fp, "%d%d", &img_height, &img_width);

  int *src_img = (int *)malloc(sizeof(int) * img_height * img_width);
  for (int i = 0; i < img_height * img_width; ++i) {
    fscanf(fp, "%d", &src_img[i]);
  }
  fclose(fp);

  int *enhancedImg = imgCLAHE(src_img, img_height, img_width);
  for (int i = 0; i < img_height*img_width; ++i)
  {
      printf("%d ", enhancedImg[i] >> 16);
  }
  for (int i = 0; i < img_height*img_width; ++i)
  {
      printf("%d ", (enhancedImg[i] >> 8) & 0x00FF);
  }
  for (int i = 0; i < img_height*img_width; ++i)
  {
      printf("%d ", enhancedImg[i] & 0x0000FF);
  }
  free(enhancedImg);
  return 0;
}
