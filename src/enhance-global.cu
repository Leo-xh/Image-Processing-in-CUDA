#include "hip/hip_runtime.h"
#include "enhance.h"
#include "common.h"
#include <stdio.h>
#include <stdlib.h>

// int updiv(int x, int y) { return (x + y - 1) / y; }
__global__ void CLAHEPre(int * hsi_img, int * g_frq, int height, int width)
// the 'tile' size is the same with the block size, 1 block for 9 tile
{
    __shared__ int frq[9*256+256];
    int lt_x = __umul24(blockIdx.x, blockDim.x*3) + threadIdx.x,
      lt_y = __umul24(blockIdx.y, blockDim.y*3) + threadIdx.y;
    int lt_idx = __umul24(lt_y, width) + lt_x;
    int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
    int per_thread = 9;
    if(thread_idx < 256) {
        for (int i = 0; i < per_thread; ++i)
        {
            frq[thread_idx*per_thread+i] = 0;
        }
    }
    if (thread_idx == 0) {
        // printf("%d %d\n", blockIdx.x, blockIdx.y);
        for (int i = 0; i < 256; ++i)
        {
            frq[9*256+i] = 0;
        }
    }
    
    __syncthreads();

    for (int i = 0; i < 3; ++i)
    {
        int tmp_x = lt_x;
        int tmp_y = lt_y + i*TILESIZE;
        int tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            atomicAdd(&frq[(i*3+0)*256+(hsi_img[tmp_idx]&0x0000FF)], 1);
        }
        tmp_x = lt_x + TILESIZE;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            atomicAdd(&frq[(i*3+1)*256+(hsi_img[tmp_idx]&0x0000FF)], 1);
        }
        tmp_x = lt_x + TILESIZE*2;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            atomicAdd(&frq[(i*3+2)*256+(hsi_img[tmp_idx]&0x0000FF)], 1);
        }
    }
    __syncthreads();

    // if(thread_idx < 256) {
    //     for (int i = 0; i < 9; ++i)
    //     {
    //         int overflow = (frq[i*256+thread_idx] > THRESHOLD)? frq[i*256+thread_idx] - THRESHOLD : 0;
    //         frq[i*256+thread_idx] -= overflow;
    //         atomicAdd(&frq[9*256+i], overflow);
    //     }
    // }
    // __syncthreads();

    // if(thread_idx < 256) {
    //     for (int i = 0; i < 9; ++i)
    //     {
    //         frq[i*256+thread_idx] += frq[9*256+i]/256;
    //     }
    // }

    __syncthreads();

    for (int i = 0; i < 9; ++i)
    {
        for (int stride = 1; stride < 256; stride <<= 1)
        {
            __syncthreads();
            int val;
            if(thread_idx < 256)
                val = (thread_idx > stride)? frq[i*256+thread_idx-stride]:0;
            __syncthreads();
            if(thread_idx < 256)
                frq[i*256+thread_idx] += val;
        }
    }

    __syncthreads();
    
    if (thread_idx < 256)
    {
        for (int i = 0; i < 9; ++i)
        {
            atomicAdd(&frq[9*256+thread_idx], frq[i*256+thread_idx]);
        }
    }

    __syncthreads();

    if (thread_idx < 256)
    {
        atomicAdd(&g_frq[thread_idx] ,frq[9*256+thread_idx]);
    }

}

__global__ void CLAHEAft(int * hsi_img, int * g_frq, int height, int width)
{
    // int over = 0;
    // int THRESHOLD = height * width / 4;
    int lt_x = __umul24(blockIdx.x, blockDim.x*3) + threadIdx.x,
      lt_y = __umul24(blockIdx.y, blockDim.y*3) + threadIdx.y;
    int lt_idx = __umul24(lt_y, width) + lt_x;
    // int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;

    // if(blockIdx.x == 0 && blockIdx.y == 0) {
    //     if(thread_idx < 256) {
    //         int overflow = (g_frq[thread_idx] > THRESHOLD)? g_frq[thread_idx] - THRESHOLD : 0;
    //         g_frq[thread_idx] -= overflow;
    //         atomicAdd(&over, overflow);
    //     }

    //     if(thread_idx < 256) {
    //         g_frq[thread_idx] += over/256;
    //     }
    // }
    for (int i = 0; i < 3; ++i)
    {
        int tmp_x = lt_x;
        int tmp_y = lt_y + i*TILESIZE;
        int tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            hsi_img[tmp_idx] = (hsi_img[tmp_idx] & 0xFFFF00) + (1.0*g_frq[(hsi_img[tmp_idx]&0x0000FF)]/(height*width))*255;
            // if ((1.0*frq[(i*3+0)*256+(hsi_img[tmp_idx]&0x0000FF)]/(TILESIZE*TILESIZE))*255 > 255) {
            //     printf("==>ERROR!\n");
            // }
        }
        tmp_x = lt_x + TILESIZE;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            hsi_img[tmp_idx] = (hsi_img[tmp_idx] & 0xFFFF00) + (1.0*g_frq[(hsi_img[tmp_idx]&0x0000FF)]/(height*width))*255;
            // if ((1.0*frq[(i*3+1)*256+(hsi_img[tmp_idx]&0x0000FF)]/(TILESIZE*TILESIZE))*255 > 255) {
            //     printf("==>ERROR!\n");
            // }
        }
        tmp_x = lt_x + TILESIZE*2;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if(tmp_x < width && tmp_y < height) {
            hsi_img[tmp_idx] = (hsi_img[tmp_idx] & 0xFFFF00) + (1.0*g_frq[(hsi_img[tmp_idx]&0x0000FF)]/(height*width))*255;
            // if ((1.0*frq[(i*3+2)*256+(hsi_img[tmp_idx]&0x0000FF)]/(TILESIZE*TILESIZE))*255 > 255) {
                // printf("==>ERROR!\n");
            // }
        }
    }

}
// bool compare(int *one, int *two, int img_height, int img_width) {
//     for (int i = 0; i < img_height*img_width; ++i)
//     {
//         if(one[i]&0xFFFF00 != two[i]&0xFFFF00) {
//             return false;
//         }
//     }
//     return true;
// }

int* imgCLAHE_Global(int *src_img, int img_height, int img_width)
{
    int * d_rgb_img, * d_hsi_img, *d_g_frq;
    int * ret_img;
    int *h_img_one, *h_img_two;
    ret_img = (int*)malloc(img_height*img_width*sizeof(int));
    h_img_one = (int*)malloc(img_height*img_width*sizeof(int));
    h_img_two = (int*)malloc(img_height*img_width*sizeof(int));
    hipMalloc((void**)& d_rgb_img, img_height*img_width*sizeof(int));
    hipMalloc((void**)& d_hsi_img, img_height*img_width*sizeof(int));
    hipMalloc((void**)& d_g_frq, 256*sizeof(int));
    hipMemset(d_g_frq, 0, 256*sizeof(int));
    hipMemcpy(d_rgb_img, src_img, img_height*img_width*sizeof(int), hipMemcpyHostToDevice);
    dim3 block(TILESIZE,TILESIZE);
    dim3 grid1(updiv(img_width, TILESIZE), updiv(img_height, TILESIZE));
    dim3 grid2(updiv(img_width, TILESIZE*3), updiv(img_height, TILESIZE*3));

    RGB2HSI<<<grid1, block>>>(d_rgb_img, d_hsi_img, img_height, img_width);

    // hipMemcpy(h_img_one, d_hsi_img, img_height*img_width*sizeof(int), hipMemcpyDeviceToHost);
    CLAHEPre<<<grid2, block>>>(d_hsi_img, d_g_frq, img_height, img_width);
    CLAHEAft<<<grid2, block>>>(d_hsi_img, d_g_frq, img_height, img_width);
    // hipMemcpy(h_img_two, d_hsi_img, img_height*img_width*sizeof(int), hipMemcpyDeviceToHost);
    // if(!compare(h_img_one, h_img_two, img_height, img_width)) {
    //     printf("===> ERROR!HSI matrix changed!\n");
    // }
    HSI2RGB<<<grid1, block>>>(d_hsi_img, d_rgb_img, img_height, img_width);
    hipMemcpy(ret_img, d_rgb_img, img_height*img_width*sizeof(int), hipMemcpyDeviceToHost);
    return ret_img;
}

int main(int argc, char **argv) {
  int img_height, img_width;

  FILE *fp;
  fp = fopen(argv[1], "r");
  fscanf(fp, "%d%d", &img_height, &img_width);

  int *src_img = (int *)malloc(sizeof(int) * img_height * img_width);
  for (int i = 0; i < img_height * img_width; ++i) {
    fscanf(fp, "%d", &src_img[i]);
  }
  fclose(fp);

  int *enhancedImg = imgCLAHE_Global(src_img, img_height, img_width);
  for (int i = 0; i < img_height*img_width; ++i)
  {
      printf("%d ", enhancedImg[i] >> 16);
  }
  for (int i = 0; i < img_height*img_width; ++i)
  {
      printf("%d ", (enhancedImg[i] >> 8) & 0x00FF);
  }
  for (int i = 0; i < img_height*img_width; ++i)
  {
      printf("%d ", enhancedImg[i] & 0x0000FF);
  }
  free(enhancedImg);
  return 0;
}
