#include "hip/hip_runtime.h"
﻿/*
 * @Author: X Wang, Y xiao, Ch Yang, G Ye
 * @Date: 2019-06-17 00:57:53
 * @Last Modified by: X Wang, Y Xiao, Ch Yang, G Ye
 * @Last Modified time: 2019-06-17 01:02:37
 * @file description:
    global histgram ehancement
 */
#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "enhance.h"

/*
hsi_img: image in HSV model
rgb_img: image in RGB model
*/

// count the histgram
__global__ void CLAHEPre(int *hsi_img, int *g_frq, int height, int width)
// the 'tile' size is the same with the block size, 1 block for 9 tile
{
  __shared__ int frq[9 * 256 + 256];
  int lt_x = __umul24(blockIdx.x, blockDim.x * 3) + threadIdx.x,
  lt_y = __umul24(blockIdx.y, blockDim.y * 3) + threadIdx.y;
  int lt_idx = __umul24(lt_y, width) + lt_x;
  int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  int per_thread = 9;
  if (thread_idx < 256) {
    for (int i = 0; i < per_thread; ++i) {
      frq[thread_idx * per_thread + i] = 0;
    }
  }
  if (thread_idx == 0) {
    for (int i = 0; i < 256; ++i) {
      frq[9 * 256 + i] = 0;
    }
  }
  
  __syncthreads();
  
  for (int i = 0; i < 3; ++i) {
    int tmp_x = lt_x;
    int tmp_y = lt_y + i * TILESIZE;
    int tmp_idx = __umul24(tmp_y, width) + tmp_x;
    if (tmp_x < width && tmp_y < height) {
      atomicAdd(&frq[(i * 3 + 0) * 256 + (hsi_img[tmp_idx] & 0x0000FF)], 1);
    }
    tmp_x = lt_x + TILESIZE;
    tmp_idx = __umul24(tmp_y, width) + tmp_x;
    if (tmp_x < width && tmp_y < height) {
      atomicAdd(&frq[(i * 3 + 1) * 256 + (hsi_img[tmp_idx] & 0x0000FF)], 1);
    }
    tmp_x = lt_x + TILESIZE * 2;
    tmp_idx = __umul24(tmp_y, width) + tmp_x;
    if (tmp_x < width && tmp_y < height) {
      atomicAdd(&frq[(i * 3 + 2) * 256 + (hsi_img[tmp_idx] & 0x0000FF)], 1);
    }
  }
  __syncthreads();
  
  for (int i = 0; i < 9; ++i) {
    for (int stride = 1; stride < 256; stride <<= 1) {
      __syncthreads();
      int val;
      if (thread_idx < 256)
      val = (thread_idx > stride) ? frq[i * 256 + thread_idx - stride] : 0;
      __syncthreads();
      if (thread_idx < 256) frq[i * 256 + thread_idx] += val;
    }
  }
  
  __syncthreads();
  
  if (thread_idx < 256) {
    for (int i = 0; i < 9; ++i) {
      atomicAdd(&frq[9 * 256 + thread_idx], frq[i * 256 + thread_idx]);
    }
  }
  
  __syncthreads();
  
  if (thread_idx < 256) {
    atomicAdd(&g_frq[thread_idx], frq[9 * 256 + thread_idx]);
  }
}

// change the intensive value of the image
__global__ void CLAHEAft(int *hsi_img, int *g_frq, int height, int width) {
  int lt_x = __umul24(blockIdx.x, blockDim.x * 3) + threadIdx.x,
      lt_y = __umul24(blockIdx.y, blockDim.y * 3) + threadIdx.y;
      int lt_idx = __umul24(lt_y, width) + lt_x;
      for (int i = 0; i < 3; ++i) {
        int tmp_x = lt_x;
        int tmp_y = lt_y + i * TILESIZE;
        int tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if (tmp_x < width && tmp_y < height) {
          int tmp =
          (1.0 * g_frq[(hsi_img[tmp_idx] & 0x0000FF)] / (height * width)) * 255;
          hsi_img[tmp_idx] =
          (hsi_img[tmp_idx] & 0xFFFF00) + ((255 < tmp) ? 255 : tmp);
        }
        tmp_x = lt_x + TILESIZE;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if (tmp_x < width && tmp_y < height) {
          int tmp =
          (1.0 * g_frq[(hsi_img[tmp_idx] & 0x0000FF)] / (height * width)) * 255;
          hsi_img[tmp_idx] =
          (hsi_img[tmp_idx] & 0xFFFF00) + ((255 < tmp) ? 255 : tmp);
        }
        tmp_x = lt_x + TILESIZE * 2;
        tmp_idx = __umul24(tmp_y, width) + tmp_x;
        if (tmp_x < width && tmp_y < height) {
          int tmp =
          (1.0 * g_frq[(hsi_img[tmp_idx] & 0x0000FF)] / (height * width)) * 255;
          hsi_img[tmp_idx] =
          (hsi_img[tmp_idx] & 0xFFFF00) + ((255 < tmp) ? 255 : tmp);
        }
      }
    }
    bool compare(int *one, int *two, int img_height, int img_width) {
      for (int i = 0; i < img_height * img_width; ++i) {
        if (one[i] & 0xFFFF00 != two[i] & 0xFFFF00) {
          return false;
    }
  }
  return true;
}

int *imgCLAHE_Global(int *src_img, int img_height, int img_width) {
  int *d_rgb_img, *d_hsi_img, *d_g_frq;
  int *ret_img;
  int *h_img_one, *h_img_two;
  ret_img = (int *)malloc(img_height * img_width * sizeof(int));
  h_img_one = (int *)malloc(img_height * img_width * sizeof(int));
  h_img_two = (int *)malloc(img_height * img_width * sizeof(int));
  hipMalloc((void **)&d_rgb_img, img_height * img_width * sizeof(int));
  hipMalloc((void **)&d_hsi_img, img_height * img_width * sizeof(int));
  hipMalloc((void **)&d_g_frq, 256 * sizeof(int));
  hipMemset(d_g_frq, 0, 256 * sizeof(int));
  hipMemset(d_hsi_img, 0, img_height * img_width * sizeof(int));
  hipMemcpy(d_rgb_img, src_img, img_height * img_width * sizeof(int),
             hipMemcpyHostToDevice);
  dim3 block(TILESIZE, TILESIZE);
  dim3 grid1(updiv(img_width, TILESIZE), updiv(img_height, TILESIZE));
  dim3 grid2(updiv(img_width, TILESIZE * 3), updiv(img_height, TILESIZE * 3));

  RGB2HSI<<<grid1, block>>>(d_rgb_img, d_hsi_img, img_height, img_width);

  CLAHEPre<<<grid2, block>>>(d_hsi_img, d_g_frq, img_height, img_width);
  CLAHEAft<<<grid2, block>>>(d_hsi_img, d_g_frq, img_height, img_width);
  
  HSI2RGB<<<grid1, block>>>(d_hsi_img, d_rgb_img, img_height, img_width);
  hipMemcpy(ret_img, d_rgb_img, img_height * img_width * sizeof(int),
             hipMemcpyDeviceToHost);

  return ret_img;
}
